#include "hip/hip_runtime.h"
#include "Header.h"
#include "math.h"


/// ��� ������� ��������� � ���� ����� ������ ����� ��������� � ����� Header.h

__device__ double get_square(const double& x1, const double& y1, const double& dx1, const double& dy1, const double& x2, const double& y2, //
    const double& dx2, const double& dy2,  double& n1, double& n2, double& dist)
{
    if (fabs(fabs(x1 - x2) - dx1 - dx2) < geo)
    {
        n1 = (x2 - x1) / fabs(x1 - x2);
        n2 = 0.0;
        dist = min(dx1, dx2);
        return 2.0 * min(dy1, dy2);
    }
    else if (fabs(fabs(y1 - y2) - dy1 - dy2) < geo)
    {
        n2 = (y2 - y1) / fabs(y1 - y2);
        n1 = 0.0;
        dist = min(dy1, dy2);
        return 2.0 * min(dx1, dx2);
    }
    else
    {
        printf("Error:  get_square: %lf, %lf, %lf, %lf, %lf,  %lf,  %lf,  %lf\n", //
            x1, y1, x2, y2, dx1, dy1, dx2, dy2);
    }
    return 0.0;
}

__device__ double HLLC_2d_Korolkov_b_s(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L,//
    const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, //
    double* P, double& PQ, const double& n1, const double& n2, const double& rad, int metod)
    // BestSeries
    // ������ ���������� 2� ���������
    //
    //  �����:
    // P[1]       // ��������
    // P[2]
    // P[0]       // �����
    // P[3]       // �������
{
    double t1 = -n2;
    double t2 = n1;

    double u1, v1, u2, v2;
    u1 = v1_L * n1 + v2_L * n2;
    v1 = v1_L * t1 + v2_L * t2;
    u2 = v1_R * n1 + v2_R * n2;
    v2 = v1_R * t1 + v2_R * t2;

    double sqrtroL = sqrt(ro_L);
    double sqrtroR = sqrt(ro_R);
    double cL = sqrt(ggg * p_L / ro_L);
    double cR = sqrt(ggg * p_R / ro_R);


    double uu_L = (kv(v1_L) + kv(v2_L)) / 2.0;
    double uu_R = (kv(v1_R) + kv(v2_R)) / 2.0;



    double SL = min(u1, u2) - max(cL, cR);
    double SR = max(u1, u2) + max(cL, cR);

    double suR = (SR - u2);
    double suL = (SL - u1);

    double SM = (suR * ro_R * u2 - suL * ro_L * u1 - p_R + p_L) //
        / (suR * ro_R - suL * ro_L);

    double PTT = (suR * ro_R * p_L - suL * ro_L * p_R + ro_L * ro_R * suR * suL * (u2 - u1)) / (suR * ro_R - suL * ro_L);

    double UU = max(fabs(SL), fabs(SR));
    double time = kurant * rad / UU;

    double FL[5], FR[5], UL[5], UR[5];

    double e1 = p_L / g1 + ro_L * uu_L;
    double e2 = p_R / g1 + ro_R * uu_R;


    FL[0] = ro_L * u1;
    FL[1] = ro_L * u1 * u1 + p_L;
    FL[2] = ro_L * u1 * v1;
    FL[3] = (e1 + p_L) * u1;
    FL[4] = Q_L * u1;

    if (SL >= 0.0)
    {
        P[1] = n1 * FL[1] + t1 * FL[2];     // ��������
        P[2] = n2 * FL[1] + t2 * FL[2];
        P[0] = FL[0];                       // �����
        P[3] = FL[3];                       // �������
        PQ = FL[4];
        return time;
    }

    FR[0] = ro_R * u2;
    FR[1] = ro_R * u2 * u2 + p_R;
    FR[2] = ro_R * u2 * v2;
    FR[3] = (e2 + p_R) * u2;
    FR[4] = Q_R * u2;

    if (SR <= 0.0)
    {
        P[1] = n1 * FR[1] + t1 * FR[2];     // ��������
        P[2] = n2 * FR[1] + t2 * FR[2];
        P[0] = FR[0];                       // �����
        P[3] = FR[3];                       // �������
        PQ = FR[4];
        return time;
    }

    UL[0] = ro_L;
    UL[1] = ro_L * u1;
    UL[2] = ro_L * v1;
    UL[3] = e1;
    UL[4] = Q_L;

    UR[0] = ro_R;
    UR[1] = ro_R * u2;
    UR[2] = ro_R * v2;
    UR[3] = e2;
    UR[4] = Q_R;

    if (metod == 0)
    {
        double  PO[5];
        for (int i = 0; i < 5; i++)
        {
            PO[i] = (SR * FL[i] - SL * FR[i] + SR * SL * (UR[i] - UL[i])) / (SR - SL);
        }

        P[1] = n1 * PO[1] + t1 * PO[2];     // ��������
        P[2] = n2 * PO[1] + t2 * PO[2];
        P[0] = PO[0];                       // �����
        P[3] = PO[3];                       // �������
        PQ = PO[4];
        return time;
    }


    double ro_LL = ro_L * (SL - u1) / (SL - SM);
    double ro_RR = ro_R * (SR - u2) / (SR - SM);
    double Q_LL = Q_L * (SL - u1) / (SL - SM);
    double Q_RR = Q_R * (SR - u2) / (SR - SM);


    double UZ0 = (SR * UR[0] - SL * UL[0] + FL[0] - FR[0]) / (SR - SL);
    double UZ1 = (SR * UR[1] - SL * UL[1] + FL[1] - FR[1]) / (SR - SL);
    double UZ2 = (SR * UR[2] - SL * UL[2] + FL[2] - FR[2]) / (SR - SL);
    double UZ3 = (SR * UR[3] - SL * UL[3] + FL[3] - FR[3]) / (SR - SL);
    double UZ4 = (SR * UR[4] - SL * UL[4] + FL[4] - FR[4]) / (SR - SL);
    double vzL, vzR, vLL, vRR, ppLR, ee1, ee2;


    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double rzR = ro_R * suRm;
    double rzL = ro_L * suLm;

    double ptzR = p_R + ro_R * suR * (SM - u2);
    double ptzL = p_L + ro_L * suL * (SM - u1);
    double ptz = (ptzR + ptzL) / 2.0;


    vRR = UZ2 / UZ0;
    vLL = vRR;


    ee2 = e2 * suRm + (ptz * SM - p_R * u2) / (SR - SM);
    ee1 = e1 * suLm + (ptz * SM - p_L * u1) / (SL - SM);


    double  ULL[5], URR[5], PO[5];
    ULL[0] = ro_LL;
    ULL[1] = ro_LL * SM;
    ULL[2] = ro_LL * vLL;
    ULL[3] = ee1;
    ULL[4] = Q_LL;

    URR[0] = ro_RR;
    URR[1] = ro_RR * SM;
    URR[2] = ro_RR * vRR;
    URR[3] = ee2;
    URR[4] = Q_RR;

    if (SL < 0.0 && SM >= 0.0)
    {
        for (int i = 0; i < 5; i++)
        {
            PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
        }
    }
    else if (SR > 0.0 && SM < 0.0)
    {
        for (int i = 0; i < 5; i++)
        {
            PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
        }
    }

    P[1] = n1 * PO[1] + t1 * PO[2];     // ��������
    P[2] = n2 * PO[1] + t2 * PO[2];
    P[0] = PO[0];                       // �����
    P[3] = PO[3];                       // �������
    PQ = PO[4];

    return time;
}

__global__ void Cuda_main_HLLDQ(int* NN, double* X, double* Y, int* Size,//
    double* RO1, double* RO2, double* Q1, double* Q2, double* P1, double* P2, double* U1, double* U2, double* V1, double* V2,//
    int* SOSED, int* L, int* R, double* T, double* T_do, int step_, double DX, double DY, int metod)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // ���������� ������ �������� ������
    if (index > * NN - 1)
    {
        return;
    }
    double x, y, ro, p, u, v, Q;
    int size = Size[index];
    double dx = (DX / pow(2, size - 1)) / 2.0;   // �������� ����� ������
    double dy = (DY / pow(2, size - 1)) / 2.0;   // �������� ������ ������
    int l = L[index];
    int r = R[index];
    x = X[index];
    y = Y[index];
    ro = RO1[index];
    p = P1[index];
    u = U1[index];
    v = V1[index];
    Q = Q1[index];
    double dist = sqrt(kv(x) + kv(y));


    if (dist < Distant) // || (ddd <= 4.0 && x > -5 && x < 0) ) //(ddd < 5.76 || ddd2 <= 2.0) //1.5
    {
        RO2[index] = ro;
        P2[index] = p;
        U2[index] = u;
        V2[index] = v;
        Q2[index] = Q;
    }
    else
    {
        double PQ = 0.0;
        double n1 = 0.0;
        double n2 = 0.0;
        double dist = 0.0;
        double P[4] = { 0.0 };
        P[0] = P[1] = P[2] = P[3] = 0.0;
        double Potok[5] = { 0.0 };
        Potok[0] = Potok[1] = Potok[2] = Potok[3] = Potok[4] = 0.0;
        double tmin = 10000000;
        double Volume = dx * dy * 4.0;
        int ii = 0;
        double x2, y2, dx2, dy2, ro2, p2, u2, v2, Q_2, size2;
        double roC = 1.0; 
        double pC = 1.0; 
        double uC = Velosity_inf;
        double vC = 0.0;
        double QC = 100.0;

        for (int i = l; i <= r; i++)
        {
            ii = SOSED[i];
            if (ii >= 0)
            {
                x2 = X[ii];
                y2 = Y[ii];
                size2 = Size[ii];
                dx2 = (DX / pow(2, size2 - 1)) / 2.0;   // �������� ����� ������
                dy2 = (DY / pow(2, size2 - 1)) / 2.0;   // �������� ������ ������
                ro2 = RO1[ii];
                p2 = P1[ii];
                u2 = U1[ii];
                v2 = V1[ii];
                Q_2 = Q1[ii];
                double S = get_square(x, y, dx, dy, x2, y2, dx2, dy2, n1, n2, dist);

                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, ro2, Q_2, p2, u2, v2, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else if (ii == -1)
            {
                double S = dy * 2.0;
                n1 = 1.0;
                n2 = 0.0;
                dist = dx;

                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, roC, QC, pC, uC, vC, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else if (ii == -2)
            {
                double S = dy * 2.0;
                n1 = -1.0;
                n2 = 0.0;
                dist = dx;

                double uu = u;
                if (uu > Velosity_inf && step_ < 300000)
                {
                    uu = Velosity_inf;
                }

                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, ro, Q, p, uu, v, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else if (ii == -3)
            {
                double S = dx * 2.0;
                n1 = 0.0;
                n2 = 1.0;
                dist = dy;

                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, ro, Q, p, u, v, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else if (ii == -4)
            {
                double S = dx * 2.0;
                n1 = 0.0;
                n2 = -1.0;
                dist = dy;

                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, ro, Q, p, u, v, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else
            {
                printf("Error 12438wedew4353jdyu. Ne doljni suda popadat = %d \n", ii);
            }
        }

        double ro3, p3, u3, v3, Q33;

        ro3 = ro - *T_do * (Potok[0] / Volume + ro * v / y);
        Q33 = Q - (*T_do / Volume) * Potok[4] - *T_do * Q * v / y;
        if (ro3 <= 0)
        {
            printf("Problemsssss  ro < 0! %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf\n", x, y, dx, dy, ro, p, u, v, Q);
            ro3 = 0.00001;
        }
        u3 = (ro * u - *T_do * (Potok[1] / Volume + ro * v * u / y)) / ro3;
        v3 = (ro * v - *T_do * (Potok[2] / Volume + ro * v * v / y)) / ro3;
        p3 = (((p / (ggg - 1) + ro * (u * u + v * v) * 0.5) - *T_do * (Potok[3] / Volume + //
            + v * (ggg * p / (ggg - 1) + ro * (u * u + v * v) * 0.5) / y)) - //
            0.5 * ro3 * (u3 * u3 + v3 * v3)) * (ggg - 1);
        if (p3 <= 0)
        {
            p3 = 0.000001;
        }

        Q2[index] = Q33;
        RO2[index] = ro3;
        P2[index] = p3;
        U2[index] = u3;
        V2[index] = v3;

        if (*T > tmin)
        {
            *T = tmin;
            __threadfence();
        }
    }

}

