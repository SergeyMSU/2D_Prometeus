#include "hip/hip_runtime.h"
#include "Header.h"
#include "math.h"


/// ��� ������� ��������� � ���� ����� ������ ����� ��������� � ����� Header.h

__global__ void Cuda_main_HLLDQ(int* NN, double* X, double* Y, int* Size,//
    double* RO1, double* RO2, double* Q1, double* Q2, double* P1, double* P2, double* U1, double* U2, double* V1, double* V2,//
    int* SOSED, int* L, int* R, double* T, double* T_do, int step_, double M_inf_, const double& DX, const double& DY, int metod = 0)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // ���������� ������ �������� ������
    if (index > * NN - 1)
    {
        return;
    }
    double x, y, z, dx, dy, ro, p, u, v, Q;
    int size = Size[index];
    double dx = (DX / pow(2, size - 1)) / 2.0;   // �������� ����� ������
    double dy = (DY / pow(2, size - 1)) / 2.0;   // �������� ������ ������
    int l = L[index];
    int r = R[index];
    x = X[index];
    y = Y[index];
    ro = RO1[index];
    p = P1[index];
    u = U1[index];
    v = V1[index];
    Q = Q1[index];

    //double ddd = kv(y) + kv(z);
    //double ddd2 = kv(x + 0.8) + kv(y) + kv(z);
    //double dist3 = kv(x + 1.0) / kv(1.6) + kv(y) / kv(1.6) + kv(z) / kv(1.6);
    double dist3 = kv(x + 1.08) / kv(2.4) + kv(y) / kv(2.0) + kv(z) / kv(2.0);

    if (dist3 < 1.0) // || (ddd <= 4.0 && x > -5 && x < 0) ) //(ddd < 5.76 || ddd2 <= 2.0) //1.5
    {
        RO2[index] = ro;
        P2[index] = p;
        U2[index] = u;
        V2[index] = v;
        Q2[index] = Q;
    }
    else
    {
        double PQ = 0.0;
        double n1 = 0.0;
        double n2 = 0.0;
        double dist = 0.0;
        double P[8] = { 0.0 };
        P[0] = P[1] = P[2] = P[3] = P[4] = P[5] = P[6] = P[7] = 0.0;
        double Potok[10] = { 0.0 };
        Potok[0] = Potok[1] = Potok[2] = Potok[3] = Potok[4] = Potok[5] = Potok[6] = Potok[7] = Potok[8] = Potok[9] = 0.0;
        double tmin = 1000;
        double Volume = dx * dy * 4.0;
        int ii = 0;
        double x2, y2, dx2, dy2, ro2, p2, u2, v2, sks, Q_2, size2;
        double roC = 1.0; // 8.2598; //  1.0;
        double pC = 1.0 / (ggg); // 1.0 / (ggg * M_inf * M_inf);
        double uC = -M_inf_; // -1.0;
        double vC = 0.0;
        double QC = 100.0;


        for (int i = l; i <= r; i++)
        {
            ii = SOSED[i];
            if (ii >= 0)
            {
                x2 = X[ii];
                y2 = Y[ii];
                size2 = Size[ii];
                double dx2 = (DX / pow(2, size - 1)) / 2.0;   // �������� ����� ������
                double dy2 = (DY / pow(2, size - 1)) / 2.0;   // �������� ������ ������
                ro2 = RO1[ii];
                p2 = P1[ii];
                u2 = U1[ii];
                v2 = V1[ii];
                Q_2 = Q1[ii];
                double S = get_square(x, y, z, dx, dy, dz, x2, y2, z2, dx2, dy2, dz2, n1, n2, n3, dist);
                if (diver == true)
                {
                    sks = n1 * (bx + bx2) / 2.0 + n2 * (by + by2) / 2.0 + n3 * (bz + bz2) / 2.0;
                }
                else
                {
                    sks = 0.0;
                }
                Potok[8] = Potok[8] + sks * S;

                if (!kor_Sol || metod == 1)//(y * y + z * z < 225 && y2 * y2 + z2 * z2 < 225 && x > -15 && x2 > -15 && x < 8 && x2 < 8  && step_ > 10000)
                {
                    tmin = min(tmin, HLLDQ_Alexashov(ro, Q, p, u, v, w, bx, by, bz, ro2, Q_2, p2, u2, v2, w2, bx2, by2, bz2, P, PQ, n1, n2, n3, dist, metod));
                }
                else
                {
                    tmin = min(tmin, HLLDQ_Korolkov(ro, Q, p, u, v, w, bx, by, bz, ro2, Q_2, p2, u2, v2, w2, bx2, by2, bz2, P, PQ, n1, n2, n3, dist, metod));
                }
                for (int k = 0; k < 8; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[9] = Potok[9] + PQ * S;
            }
            else if (ii == -1)
            {
                double S = dy * dz * 4.0;
                n1 = 1.0;
                n2 = 0.0;
                n3 = 0.0;
                dist = dx;
                if (diver == true)
                {
                    sks = n1 * (bx + bxC) / 2.0 + n2 * (by + byC) / 2.0 + n3 * (bz + bzC) / 2.0;
                }
                else
                {
                    sks = 0.0;
                }
                /*double uu = u;
                if (uu < 0.0)
                {
                    uu = 0.0;
                }*/
                Potok[8] = Potok[8] + sks * S;
                if (!kor_Sol || metod == 1 || metod == 3)
                {
                    tmin = min(tmin, HLLDQ_Alexashov(ro, Q, p, u, v, w, bx, by, bz, roC, QC, pC, uC, vC, wC, bxC, byC, bzC, P, PQ, n1, n2, n3, dist, metod));
                }
                else
                {
                    tmin = min(tmin, HLLDQ_Korolkov(ro, Q, p, u, v, w, bx, by, bz, roC, QC, pC, uC, vC, wC, bxC, byC, bzC, P, PQ, n1, n2, n3, dist, metod));
                }
                //  ����� ������� �������� ������ ��� ��������� �������
                /*double b2R = kv(bxC) + kv(byC) + kv(bzC);
                double ptR = pC + b2R / 2.0;
                double upt2 = (kv(uC) + kv(vC) + kv(wC)) / 2.0;
                double sbv2 = uC * bxC + vC * byC + wC * bzC;
                double e2 = pC / g1 + roC * upt2 + b2R / 2.0;

                P[0] = roC * uC;
                P[1] = roC * uC * uC + ptR - kv(bxC);
                P[2] = roC * uC * vC - bxC * byC;
                P[3] = roC * uC * wC - bxC * bzC;
                P[7] = (e2 + ptR) * uC - bxC * sbv2;
                P[4] = 0.0;
                P[5] = uC * byC - vC * bxC;
                P[6] = uC * bzC - wC * bxC;*/

                for (int k = 0; k < 8; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[9] = Potok[9] + PQ * S;
            }
            else if (ii == -2)
            {
                double S = dy * dz * 4.0;
                n1 = -1.0;
                n2 = 0.0;
                n3 = 0.0;
                dist = dx;
                if (diver == true)
                {
                    sks = n1 * bx + n2 * by + n3 * bz;
                }
                else
                {
                    sks = 0.0;
                }
                Potok[8] = Potok[8] + sks * S;
                double uu = u;
                if (uu > -M_inf_ && step_ < 5000)
                {
                    uu = -M_inf_;
                }
                else if (uu > -0.01)
                {
                    uu = -0.01;
                }

                if (!kor_Sol || metod == 1 || metod == 3)
                {
                    tmin = min(tmin, HLLDQ_Alexashov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, p, uu, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }
                else
                {
                    tmin = min(tmin, HLLDQ_Korolkov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, p, uu, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }

                /*double t1, t2, t3, m1, m2, m3;
                double bx_L = bx / spi4;
                double by_L = by / spi4;
                double bz_L = bz / spi4;
                t1 = 0.0;
                t2 = 0.0;
                t3 = 1.0;
                m1 = 0.0;
                m2 = 1.0;
                m3 = 0.0;
                double u1 = uu * n1 + v * n2 + w * n3;
                double v1 = uu * t1 + v * t2 + w * t3;
                double w1 = uu * m1 + v * m2 + w * m3;
                double bn1, bt1, bm1;
                bn1 = bx_L * n1 + by_L * n2 + bz_L * n3;
                bt1 = bx_L * t1 + by_L * t2 + bz_L * t3;
                bm1 = bx_L * m1 + by_L * m2 + bz_L * m3;
                double uu_L = (kv(uu) + kv(v) + kv(w)) / 2.0;
                double bb_L = kv(bx_L) + kv(by_L) + kv(bz_L);
                double e1 = p / g1 + ro * uu_L + bb_L / 2.0;
                double pTL = p + bb_L / 2.0;

                double PO[9];

                PO[0] = ro * u1;
                PO[1] = ro * u1 * u1 + pTL - kv(bn1);
                PO[2] = ro * u1 * v1 - bn1 * bt1;
                PO[3] = ro * u1 * w1 - bn1 * bm1;
                PO[4] = (e1 + pTL) * u1 - bn1 * (u1 * bn1 + v1 * bt1 + w1 * bm1);
                PO[5] = 0.0;
                PO[6] = u1 * bt1 - v1 * bn1;
                PO[7] = u1 * bm1 - w1 * bn1;
                PO[8] = Q * u1;


                P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
                P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
                P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
                P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
                P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
                P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
                P[0] = PO[0];
                P[4] = PO[4];
                PQ = PO[8];

                double SWAP = P[4];
                P[4] = P[5];
                P[5] = P[6];
                P[6] = P[7];
                P[7] = SWAP;*/

                for (int k = 0; k < 8; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[9] = Potok[9] + PQ * S;
            }
            else if (ii == -3)
            {
                double S = dx * dz * 4.0;
                n1 = 0.0;
                n2 = 1.0;
                n3 = 0.0;
                dist = dy;
                double uu = v;
                if (uu < 0.0)
                {
                    uu = 0.0;
                }
                if (diver == true)
                {
                    sks = n1 * (bx + bx) / 2.0 + n2 * (by + by) / 2.0 + n3 * (bz + bz) / 2.0;
                }
                else
                {
                    sks = 0.0;
                }
                Potok[8] = Potok[8] + sks * S;
                if (!kor_Sol || metod == 1 || metod == 3)
                {
                    tmin = min(tmin, HLLDQ_Alexashov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, pC, u, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }
                else
                {
                    tmin = min(tmin, HLLDQ_Korolkov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, pC, u, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }
                for (int k = 0; k < 8; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[9] = Potok[9] + PQ * S;
            }
            else if (ii == -4)
            {
                double S = dx * dz * 4.0;
                n1 = 0.0;
                n2 = -1.0;
                n3 = 0.0;
                dist = dy;
                if (diver == true)
                {
                    sks = n1 * (bx + bx) / 2.0 + n2 * (by + by) / 2.0 + n3 * (bz + bz) / 2.0;
                    //sks =  n2 * (by + by) / 2.0;
                }
                else
                {
                    sks = 0.0;
                }

                Potok[8] = Potok[8] + sks * S;
                if (!kor_Sol || metod == 1 || metod == 3)
                {
                    //tmin = min(tmin, HLLDQ_Alexashov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, p, u, -v, w, -bx, by, -bz, P, PQ, n1, n2, n3, dist, metod));
                    tmin = min(tmin, HLLDQ_Alexashov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, p, u, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }
                else
                {
                    //tmin = min(tmin, HLLDQ_Korolkov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, p, u, -v, w, -bx, by, -bz, P, PQ, n1, n2, n3, dist, metod));
                    tmin = min(tmin, HLLDQ_Korolkov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, p, u, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }

                for (int k = 0; k < 8; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[9] = Potok[9] + PQ * S;
            }
            else if (ii == -5)
            {
                double S = dy * dx * 4.0;
                n1 = 0.0;
                n2 = 0.0;
                n3 = 1.0;
                dist = dz;
                if (diver == true)
                {
                    sks = n1 * (bx + bx) / 2.0 + n2 * (by + by) / 2.0 + n3 * (bz + bz) / 2.0;
                }
                else
                {
                    sks = 0.0;
                }


                Potok[8] = Potok[8] + sks * S;
                if (!kor_Sol || metod == 1 || metod == 3)
                {
                    tmin = min(tmin, HLLDQ_Alexashov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, pC, u, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }
                else
                {
                    tmin = min(tmin, HLLDQ_Korolkov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, pC, u, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }

                for (int k = 0; k < 8; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[9] = Potok[9] + PQ * S;
            }
            else if (ii == -6)
            {
                double S = dy * dx * 4.0;
                n1 = 0.0;
                n2 = 0.0;
                n3 = -1.0;
                dist = dz;
                if (diver == true)
                {
                    //sks = n1 * (bx + bx) / 2.0 + n2 * (by + by) / 2.0 + n3 * (bz + bz) / 2.0;
                    sks = n1 * (bx + bx) / 2.0 + n2 * (by + by) / 2.0;
                }
                else
                {
                    sks = 0.0;
                }


                Potok[8] = Potok[8] + sks * S;
                if (!kor_Sol || metod == 1 || metod == 3)
                {
                    tmin = min(tmin, HLLDQ_Alexashov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, p, u, v, -w, bx, by, -bz, P, PQ, n1, n2, n3, dist, metod));
                    //tmin = min(tmin, HLLDQ_Alexashov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, pC, u, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }
                else
                {
                    tmin = min(tmin, HLLDQ_Korolkov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, p, u, v, -w, bx, by, -bz, P, PQ, n1, n2, n3, dist, metod));
                    //tmin = min(tmin, HLLDQ_Korolkov(ro, Q, p, u, v, w, bx, by, bz, ro, Q, pC, u, v, w, bx, by, bz, P, PQ, n1, n2, n3, dist, metod));
                }
                for (int k = 0; k < 8; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[9] = Potok[9] + PQ * S;
            }
            else
            {
                printf("Error 12438jdyu. Ne doljni suda popadat = %d \n", ii);
            }
        }

        double ro3, p3, u3, v3, w3, bx3, by3, bz3, Q33;

        Q33 = Q - *T_do * Potok[9] / Volume;
        ro3 = ro - *T_do * Potok[0] / Volume;
        if (ro3 <= 0.0)
        {
            printf("ERROR -  dssdbfhfshjskfutytqqazz\n");
            printf("%lf, %lf, %lf, %lf\n", x, y, z, ro3);
            ro3 = ro;
        }
        u3 = (ro * u - *T_do * (Potok[1] + (bx / cpi4) * Potok[8]) / Volume) / ro3;
        v3 = (ro * v - *T_do * (Potok[2] + (by / cpi4) * Potok[8]) / Volume) / ro3;
        w3 = (ro * w - *T_do * (Potok[3] + (bz / cpi4) * Potok[8]) / Volume) / ro3;
        bx3 = (bx - *T_do * (Potok[4] + u * Potok[8]) / Volume);
        by3 = (by - *T_do * (Potok[5] + v * Potok[8]) / Volume);
        bz3 = (bz - *T_do * (Potok[6] + w * Potok[8]) / Volume);
        p3 = ((U8(ro, p, u, v, w, bx, by, bz) - *T_do * (Potok[7] + (skk(u, v, w, bx, by, bz) / cpi4) * Potok[8])//
            / Volume) - 0.5 * ro3 * kvv(u3, v3, w3) - kvv(bx3, by3, bz3) / cpi8) * (ggg - 1.0);
        //u3 = (ro * u - *T_do * (Potok[1] + (bx) * Potok[8]) / Volume) / ro3;
        //v3 = (ro * v - *T_do * (Potok[2] + (by) * Potok[8]) / Volume) / ro3;
        //w3 = (ro * w - *T_do * (Potok[3] + (bz) * Potok[8]) / Volume) / ro3;
        //bx3 = (bx - *T_do * (Potok[4] + u * Potok[8]) / Volume);
        //by3 = (by - *T_do * (Potok[5] + v * Potok[8]) / Volume);
        //bz3 = (bz - *T_do * (Potok[6] + w * Potok[8]) / Volume);
        //p3 = ((U8(ro, p, u, v, w, bx, by, bz) - *T_do * (Potok[7] + (skk(u, v, w, bx, by, bz)) * Potok[8])//
        //    / Volume) - 0.5 * ro3 * kvv(u3, v3, w3) - kvv(bx3, by3, bz3)) * (ggg - 1.0);
        if (p3 <= 0)
        {
            p3 = 0.000001;
        }

        Q2[index] = Q33;
        RO2[index] = ro3;
        P2[index] = p3;
        U2[index] = u3;
        V2[index] = v3;
        W2[index] = w3;
        /*if (Q33 / ro3 > 50)
        {
            BX2[index] = 0.0;
            BY2[index] = 0.0;
            BZ2[index] = 0.0;
        }
        else
        {
            BX2[index] = bx3;
            BY2[index] = by3;
            BZ2[index] = bz3;
        }*/
        BX2[index] = bx3;
        BY2[index] = by3;
        BZ2[index] = bz3;

        if (*T > tmin)
        {
            *T = tmin;
            __threadfence();
        }
    }

}

