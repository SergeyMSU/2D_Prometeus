#include "hip/hip_runtime.h"
#include "Header.h"
#include "math.h"


/// ��� ������� ��������� � ���� ����� ������ ����� ��������� � ����� Header.h
__device__ double polar_angle(double x, double y)
{
    if (x < 0)
    {
        return atan(y / x) + 1.0 * PI;
    }
    else if (x > 0 && y >= 0)
    {
        return atan(y / x);
    }
    else if (x > 0 && y < 0)
    {
        return atan(y / x) + 2.0 * PI;
    }
    else if (y > 0 && x >= 0 && x <= 0)
    {
        return PI / 2.0;
    }
    else if (y < 0 && x >= 0 && x <= 0)
    {
        return  3.0 * PI / 2.0;
    }
    return 0.0;
}

__device__ double polar_perenos(const double& x1, const double& y1, const double& x2, const double& y2, double& u, double& v)
{
    double phi1 = polar_angle(x1, y1);
    double phi2 = polar_angle(x2, y2);
    double fr = u * cos(phi1) + v * sin(phi1);
    double ff = -u * sin(phi1) + v * cos(phi1);
    u = fr * cos(phi2) - ff * sin(phi2);
    v = fr * sin(phi2) + ff * cos(phi2);
}

__device__ double get_square(const double& x1, const double& y1, const double& dx1, const double& dy1, const double& x2, const double& y2, //
    const double& dx2, const double& dy2,  double& n1, double& n2, double& dist)
{
    if (fabs(fabs(x1 - x2) - dx1 - dx2) < geo)
    {
        n1 = (x2 - x1) / fabs(x1 - x2);
        n2 = 0.0;
        dist = min(dx1, dx2);
        return 2.0 * min(dy1, dy2);
    }
    else if (fabs(fabs(y1 - y2) - dy1 - dy2) < geo)
    {
        n2 = (y2 - y1) / fabs(y1 - y2);
        n1 = 0.0;
        dist = min(dy1, dy2);
        return 2.0 * min(dx1, dx2);
    }
    else
    {
        printf("Error:  get_square: %lf, %lf, %lf, %lf, %lf,  %lf,  %lf,  %lf\n", //
            x1, y1, x2, y2, dx1, dy1, dx2, dy2);
    }
    return 0.0;
}

__device__ double HLLC_2d_Korolkov_b_s(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L,//
    const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, //
    double* P, double& PQ, const double& n1, const double& n2, const double& rad, int metod)
    // BestSeries
    // ������ ���������� 2� ���������
    //
    //  �����:
    // P[1]       // ��������
    // P[2]
    // P[0]       // �����
    // P[3]       // �������
{
    double t1 = -n2;
    double t2 = n1;

    double u1, v1, u2, v2;
    u1 = v1_L * n1 + v2_L * n2;
    v1 = v1_L * t1 + v2_L * t2;
    u2 = v1_R * n1 + v2_R * n2;
    v2 = v1_R * t1 + v2_R * t2;

    double sqrtroL = sqrt(ro_L);
    double sqrtroR = sqrt(ro_R);
    double cL = sqrt(ggg * p_L / ro_L);
    double cR = sqrt(ggg * p_R / ro_R);


    double uu_L = (kv(v1_L) + kv(v2_L)) / 2.0;
    double uu_R = (kv(v1_R) + kv(v2_R)) / 2.0;



    double SL = min(u1, u2) - max(cL, cR);
    double SR = max(u1, u2) + max(cL, cR);

    double suR = (SR - u2);
    double suL = (SL - u1);

    double SM = (suR * ro_R * u2 - suL * ro_L * u1 - p_R + p_L) //
        / (suR * ro_R - suL * ro_L);

    double PTT = (suR * ro_R * p_L - suL * ro_L * p_R + ro_L * ro_R * suR * suL * (u2 - u1)) / (suR * ro_R - suL * ro_L);

    double UU = max(fabs(SL), fabs(SR));
    double time = kurant * rad / UU;

    double FL[5], FR[5], UL[5], UR[5];

    double e1 = p_L / g1 + ro_L * uu_L;
    double e2 = p_R / g1 + ro_R * uu_R;


    FL[0] = ro_L * u1;
    FL[1] = ro_L * u1 * u1 + p_L;
    FL[2] = ro_L * u1 * v1;
    FL[3] = (e1 + p_L) * u1;
    FL[4] = Q_L * u1;

    if (SL >= 0.0)
    {
        P[1] = n1 * FL[1] + t1 * FL[2];     // ��������
        P[2] = n2 * FL[1] + t2 * FL[2];
        P[0] = FL[0];                       // �����
        P[3] = FL[3];                       // �������
        PQ = FL[4];
        return time;
    }

    FR[0] = ro_R * u2;
    FR[1] = ro_R * u2 * u2 + p_R;
    FR[2] = ro_R * u2 * v2;
    FR[3] = (e2 + p_R) * u2;
    FR[4] = Q_R * u2;

    if (SR <= 0.0)
    {
        P[1] = n1 * FR[1] + t1 * FR[2];     // ��������
        P[2] = n2 * FR[1] + t2 * FR[2];
        P[0] = FR[0];                       // �����
        P[3] = FR[3];                       // �������
        PQ = FR[4];
        return time;
    }

    UL[0] = ro_L;
    UL[1] = ro_L * u1;
    UL[2] = ro_L * v1;
    UL[3] = e1;
    UL[4] = Q_L;

    UR[0] = ro_R;
    UR[1] = ro_R * u2;
    UR[2] = ro_R * v2;
    UR[3] = e2;
    UR[4] = Q_R;

    if (metod == 0)
    {
        double  PO[5];
        for (int i = 0; i < 5; i++)
        {
            PO[i] = (SR * FL[i] - SL * FR[i] + SR * SL * (UR[i] - UL[i])) / (SR - SL);
        }

        P[1] = n1 * PO[1] + t1 * PO[2];     // ��������
        P[2] = n2 * PO[1] + t2 * PO[2];
        P[0] = PO[0];                       // �����
        P[3] = PO[3];                       // �������
        PQ = PO[4];
        return time;
    }


    double ro_LL = ro_L * (SL - u1) / (SL - SM);
    double ro_RR = ro_R * (SR - u2) / (SR - SM);
    double Q_LL = Q_L * (SL - u1) / (SL - SM);
    double Q_RR = Q_R * (SR - u2) / (SR - SM);


    double UZ0 = (SR * UR[0] - SL * UL[0] + FL[0] - FR[0]) / (SR - SL);
    double UZ1 = (SR * UR[1] - SL * UL[1] + FL[1] - FR[1]) / (SR - SL);
    double UZ2 = (SR * UR[2] - SL * UL[2] + FL[2] - FR[2]) / (SR - SL);
    double UZ3 = (SR * UR[3] - SL * UL[3] + FL[3] - FR[3]) / (SR - SL);
    double UZ4 = (SR * UR[4] - SL * UL[4] + FL[4] - FR[4]) / (SR - SL);
    double vzL, vzR, vLL, vRR, ppLR, ee1, ee2;


    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double rzR = ro_R * suRm;
    double rzL = ro_L * suLm;

    double ptzR = p_R + ro_R * suR * (SM - u2);
    double ptzL = p_L + ro_L * suL * (SM - u1);
    double ptz = (ptzR + ptzL) / 2.0;


    if( fabs(v1 - v2) > 0.1)
    {
        vLL = v1;
        vRR = v2;
    }
    else
    {
        vRR = UZ2 / UZ0;
        vLL = vRR;
    }


    ee2 = e2 * suRm + (ptz * SM - p_R * u2) / (SR - SM);
    ee1 = e1 * suLm + (ptz * SM - p_L * u1) / (SL - SM);


    double  ULL[5], URR[5], PO[5];
    ULL[0] = ro_LL;
    ULL[1] = ro_LL * SM;
    ULL[2] = ro_LL * vLL;
    ULL[3] = ee1;
    ULL[4] = Q_LL;

    URR[0] = ro_RR;
    URR[1] = ro_RR * SM;
    URR[2] = ro_RR * vRR;
    URR[3] = ee2;
    URR[4] = Q_RR;

    if (SL < 0.0 && SM >= 0.0)
    {
        for (int i = 0; i < 5; i++)
        {
            PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
        }
    }
    else if (SR > 0.0 && SM < 0.0)
    {
        for (int i = 0; i < 5; i++)
        {
            PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
        }
    }

    P[1] = n1 * PO[1] + t1 * PO[2];     // ��������
    P[2] = n2 * PO[1] + t2 * PO[2];
    P[0] = PO[0];                       // �����
    P[3] = PO[3];                       // �������
    PQ = PO[4];

    return time;
}

__global__ void Cuda_main_HLLDQ(int* NN, double* X, double* Y, int* Size,//
    double* RO1, double* RO2, double* Q1, double* Q2, double* P1, double* P2, double* U1, double* U2, double* V1, double* V2,//
    int* SOSED, int* L, int* R, double* T, double* T_do, int step_, double DX, double DY, int metod)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // ���������� ������ �������� ������
    if (index > * NN - 1)
    {
        return;
    }
    double x, y, ro, p, u, v, Q;
    int size = Size[index];
    double dx = (DX / pow(2, size - 1)) / 2.0;   // �������� ����� ������
    double dy = (DY / pow(2, size - 1)) / 2.0;   // �������� ������ ������
    int l = L[index];
    int r = R[index];
    x = X[index];
    y = Y[index];
    ro = RO1[index];
    p = P1[index];
    u = U1[index];
    v = V1[index];
    Q = Q1[index];
    double radius = sqrt(kv(x) + kv(y));


    if (radius <= Distant) // || (ddd <= 4.0 && x > -5 && x < 0) ) //(ddd < 5.76 || ddd2 <= 2.0) //1.5
    {
        RO2[index] = ro;
        P2[index] = p;
        U2[index] = u;
        V2[index] = v;
        Q2[index] = Q;
    }
    else
    {
        double PQ = 0.0;
        double n1 = 0.0;
        double n2 = 0.0;
        double dist = 0.0;
        double P[4] = { 0.0 };
        P[0] = P[1] = P[2] = P[3] = 0.0;
        double Potok[5] = { 0.0 };
        Potok[0] = Potok[1] = Potok[2] = Potok[3] = Potok[4] = 0.0;
        double tmin = 10000000;
        double Volume = dx * dy * 4.0;
        int ii = 0;
        double x2, y2, dx2, dy2, ro2, p2, u2, v2, Q_2, size2;
        double roC = 1.0; 
        double pC = 1.0; 
        double uC = Velosity_inf;
        double vC = 0.0;
        double QC = 100.0;
        double u1_polar, v1_polar;

        for (int i = l; i <= r; i++)
        {
            ii = SOSED[i];
            if (ii >= 0)
            {
                x2 = X[ii];
                y2 = Y[ii];
                size2 = Size[ii];
                dx2 = (DX / pow(2, size2 - 1)) / 2.0;   // �������� ����� ������
                dy2 = (DY / pow(2, size2 - 1)) / 2.0;   // �������� ������ ������
                ro2 = RO1[ii];
                p2 = P1[ii];
                u2 = U1[ii];
                v2 = V1[ii];
                Q_2 = Q1[ii];
                double S = get_square(x, y, dx, dy, x2, y2, dx2, dy2, n1, n2, dist);

                u1_polar = u;
                v1_polar = v;

                if (radius < 100)
                {
                    polar_perenos(x, y, x + n1 * dx, y + n2 * dy, u1_polar, v1_polar);
                    polar_perenos(x2, y2, x2 - n1 * dx2, y2 - n2 * dy2, u2, v2);
                }
                

                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u1_polar, v1_polar, ro2, Q_2, p2, u2, v2, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else if (ii == -1)
            {
                double S = dy * 2.0;
                n1 = 1.0;
                n2 = 0.0;
                dist = dx;

                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, roC, QC, pC, uC, vC, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else if (ii == -2)
            {
                double S = dy * 2.0;
                n1 = -1.0;
                n2 = 0.0;
                dist = dx;

                double uu = u;
                if (uu > Velosity_inf && y < 300)
                {
                    uu = Velosity_inf;
                }

                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, ro, Q, p, uu, v, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else if (ii == -3)
            {
                double S = dx * 2.0;
                n1 = 0.0;
                n2 = 1.0;
                dist = dy;

                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, ro, Q, p, u, v, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else if (ii == -4)
            {
                double S = dx * 2.0;
                n1 = 0.0;
                n2 = -1.0;
                dist = dy;

                u1_polar = u;
                v1_polar = v;

                if (radius < 100)
                {
                    u1_polar = u;
                    v1_polar = v;
                    polar_perenos(x, y, x + n1 * dx, y + n2 * dy, u1_polar, v1_polar);
                }


                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u1_polar, v1_polar, ro, Q, p, //
                                                        u1_polar, -v1_polar, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
            else
            {
                printf("Error 12438wedew4353jdyu. Ne doljni suda popadat = %d \n", ii);
            }
        }

        double ro3, p3, u3, v3, Q33;

        ro3 = ro - *T_do * (Potok[0] / Volume + ro * v / y);
        Q33 = Q - (*T_do / Volume) * Potok[4] - *T_do * Q * v / y;
        if (ro3 <= 0)
        {
            printf("Problemsssss  ro < 0! %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf\n", x, y, dx, dy, ro, p, u, v, Q);
            ro3 = 0.00001;
        }
        u3 = (ro * u - *T_do * (Potok[1] / Volume + ro * v * u / y)) / ro3;
        v3 = (ro * v - *T_do * (Potok[2] / Volume + ro * v * v / y)) / ro3;
        p3 = (((p / (ggg - 1) + ro * (u * u + v * v) * 0.5) - *T_do * (Potok[3] / Volume + //
            + v * (ggg * p / (ggg - 1) + ro * (u * u + v * v) * 0.5) / y)) - //
            0.5 * ro3 * (u3 * u3 + v3 * v3)) * (ggg - 1);
        if (p3 <= 0)
        {
            p3 = 0.000001;
        }

        Q2[index] = Q33;
        RO2[index] = ro3;
        P2[index] = p3;
        U2[index] = u3;
        V2[index] = v3;

        if (*T > tmin)
        {
            *T = tmin;
            __threadfence();
        }
    }

}

__global__ void Cuda_main_5_komponent(int* NN, double* X, double* Y, int* Size,//
    double* RO1, double* RO2, double* Q1, double* Q2, double* P1, double* P2, double* U1, double* U2, double* V1, double* V2,//
    double* RO1_H1, double* RO2_H1, double* P1_H1, double* P2_H1, double* U1_H1, double* U2_H1, double* V1_H1, double* V2_H1,//
    double* RO1_H2, double* RO2_H2, double* P1_H2, double* P2_H2, double* U1_H2, double* U2_H2, double* V1_H2, double* V2_H2,//
    double* RO1_H3, double* RO2_H3, double* P1_H3, double* P2_H3, double* U1_H3, double* U2_H3, double* V1_H3, double* V2_H3,//
    double* RO1_H4, double* RO2_H4, double* P1_H4, double* P2_H4, double* U1_H4, double* U2_H4, double* V1_H4, double* V2_H4,//
    int* SOSED, int* L, int* R, double* T, double* T_do, int step_, double DX, double DY, int metod)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // ���������� ������ �������� ������
    if (index > * NN - 1)
    {
        return;
    }
    double x, y, ro, p, u, v, Q;
    double ro_H1, p_H1, u_H1, v_H1;
    double ro_H2, p_H2, u_H2, v_H2;
    double ro_H3, p_H3, u_H3, v_H3;
    double ro_H4, p_H4, u_H4, v_H4;
    int size = Size[index];
    double dx = (DX / pow(2, size - 1)) / 2.0;   // �������� ����� ������
    double dy = (DY / pow(2, size - 1)) / 2.0;   // �������� ������ ������
    int l = L[index];
    int r = R[index];
    x = X[index];
    y = Y[index];
    ro = RO1[index];
    p = P1[index];
    u = U1[index];
    v = V1[index];
    ro_H1 = RO1_H1[index];
    p_H1 = P1_H1[index];
    u_H1 = U1_H1[index];
    v_H1 = V1_H1[index];
    ro_H2 = RO1_H2[index];
    p_H2 = P1_H2[index];
    u_H2 = U1_H2[index];
    v_H2 = V1_H2[index];
    ro_H3 = RO1_H3[index];
    p_H3 = P1_H3[index];
    u_H3 = U1_H3[index];
    v_H3 = V1_H3[index];
    ro_H4 = RO1_H4[index];
    p_H4 = P1_H4[index];
    u_H4 = U1_H4[index];
    v_H4 = V1_H4[index];
    Q = Q1[index];

    double radius = sqrt(kv(x) + kv(y));


    double PQ = 0.0;
    double n1 = 0.0;
    double n2 = 0.0;
    double dist = 0.0;
    double P[4] = { 0.0 };
    P[0] = P[1] = P[2] = P[3] = 0.0;
    double Potok[5] = { 0.0 };
    Potok[0] = Potok[1] = Potok[2] = Potok[3] = Potok[4] = 0.0;
    double Potok_H1[4] = { 0.0 };
    Potok_H1[0] = Potok_H1[1] = Potok_H1[2] = Potok_H1[3] = 0.0;
    double Potok_H2[4] = { 0.0 };
    Potok_H2[0] = Potok_H2[1] = Potok_H2[2] = Potok_H2[3] = 0.0;
    double Potok_H3[4] = { 0.0 };
    Potok_H3[0] = Potok_H3[1] = Potok_H3[2] = Potok_H3[3] = 0.0;
    double Potok_H4[4] = { 0.0 };
    Potok_H4[0] = Potok_H4[1] = Potok_H4[2] = Potok_H4[3] = 0.0;
    double tmin = 10000000;
    double Volume = dx * dy * 4.0;
    int ii = 0;
    double x2, y2, dx2, dy2, ro2, p2, u2, v2, Q_2, size2;
    double ro2_H1, p2_H1, u2_H1, v2_H1;
    double ro2_H2, p2_H2, u2_H2, v2_H2;
    double ro2_H3, p2_H3, u2_H3, v2_H3;
    double ro2_H4, p2_H4, u2_H4, v2_H4;
    double roC = 1.0;
    double pC = 1.0;
    double uC = Velosity_inf;
    double vC = 0.0;
    double QC = 100.0;
    double u1_polar, v1_polar;

    for (int i = l; i <= r; i++)
    {
        ii = SOSED[i];
        if (ii >= 0)
        {
            x2 = X[ii];
            y2 = Y[ii];
            size2 = Size[ii];
            dx2 = (DX / pow(2, size2 - 1)) / 2.0;   // �������� ����� ������
            dy2 = (DY / pow(2, size2 - 1)) / 2.0;   // �������� ������ ������
            ro2 = RO1[ii];
            p2 = P1[ii];
            u2 = U1[ii];
            v2 = V1[ii];
            ro2_H1 = RO1_H1[ii];
            p2_H1 = P1_H1[ii];
            u2_H1 = U1_H1[ii];
            v2_H1 = V1_H1[ii];
            ro2_H2 = RO1_H2[ii];
            p2_H2 = P1_H2[ii];
            u2_H2 = U1_H2[ii];
            v2_H2 = V1_H2[ii];
            ro2_H3 = RO1_H3[ii];
            p2_H3 = P1_H3[ii];
            u2_H3 = U1_H3[ii];
            v2_H3 = V1_H3[ii];
            ro2_H4 = RO1_H4[ii];
            p2_H4 = P1_H4[ii];
            u2_H4 = U1_H4[ii];
            v2_H4 = V1_H4[ii];
            Q_2 = Q1[ii];
            double S = get_square(x, y, dx, dy, x2, y2, dx2, dy2, n1, n2, dist);

            u1_polar = u;
            v1_polar = v;

            if (radius < 100)
            {
                polar_perenos(x, y, x + n1 * dx, y + n2 * dy, u1_polar, v1_polar);
                polar_perenos(x2, y2, x2 - n1 * dx2, y2 - n2 * dy2, u2, v2);
            }

            if (radius > Distant)
            {
                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H1, 1.0, p_H1, u_H1, v_H1, ro2_H1, 1.0, p2_H1, u2_H1, v2_H1, P, PQ, n1, n2, dist, metod));
                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok_H1[k] = Potok_H1[k] + P[k] * S;
                }
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H2, 1.0, p_H2, u_H2, v_H2, ro2_H2, 1.0, p2_H2, u2_H2, v2_H2, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H2[k] = Potok_H2[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H3, 1.0, p_H3, u_H3, v_H3, ro2_H3, 1.0, p2_H3, u2_H3, v2_H3, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H3[k] = Potok_H3[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H4, 1.0, p_H4, u_H4, v_H4, ro2_H4, 1.0, p2_H4, u2_H4, v2_H4, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H4[k] = Potok_H4[k] + P[k] * S;
            }

            if (radius > Distant)
            {
                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u1_polar, v1_polar, ro2, Q_2, p2, u2, v2, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
        }
        else if (ii == -1)
        {
            double S = dy * 2.0;
            n1 = 1.0;
            n2 = 0.0;
            dist = dx;

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H1, 1.0, p_H1, u_H1, v_H1, ro_H1, 1.0, p_H1, u_H1, v_H1, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H1[k] = Potok_H1[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H2, 1.0, p_H2, u_H2, v_H2, ro_H2, 1.0, p_H2, u_H2, v_H2, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H2[k] = Potok_H2[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H3, 1.0, p_H3, u_H3, v_H3, ro_H3, 1.0, p_H3, u_H3, v_H3, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H3[k] = Potok_H3[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H4, 1.0, p_H4, u_H4, v_H4, roC, 1.0, 0.5 * pC, Velosity_inf, 0.0, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H4[k] = Potok_H4[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, roC, QC, pC, uC, vC, P, PQ, n1, n2, dist, metod));

            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok[k] = Potok[k] + P[k] * S;
            }
            Potok[4] = Potok[4] + PQ * S;
        }
        else if (ii == -2)
        {
            double S = dy * 2.0;
            n1 = -1.0;
            n2 = 0.0;
            dist = dx;

            double uu = u;
            if (uu > Velosity_inf && y < 300)
            {
                uu = Velosity_inf;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H1, 1.0, p_H1, u_H1, v_H1, ro_H1, 1.0, p_H1, u_H1, v_H1, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H1[k] = Potok_H1[k] + P[k] * S;
            }
            

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H2, 1.0, p_H2, u_H2, v_H2, ro_H2, 1.0, p_H2, u_H2, v_H2, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H2[k] = Potok_H2[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H3, 1.0, p_H3, u_H3, v_H3, ro_H3, 1.0, p_H3, u_H3, v_H3, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H3[k] = Potok_H3[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H4, 1.0, p_H4, u_H4, v_H4, ro_H4, 1.0, p_H4, u_H4, v_H4, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H4[k] = Potok_H4[k] + P[k] * S;
            }


            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, ro, Q, p, uu, v, P, PQ, n1, n2, dist, metod));

            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok[k] = Potok[k] + P[k] * S;
            }
            Potok[4] = Potok[4] + PQ * S;
            
        }
        else if (ii == -3)
        {
            double S = dx * 2.0;
            n1 = 0.0;
            n2 = 1.0;
            dist = dy;

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H1, 1.0, p_H1, u_H1, v_H1, ro_H1, 1.0, p_H1, u_H1, v_H1, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H1[k] = Potok_H1[k] + P[k] * S;
            }
            

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H2, 1.0, p_H2, u_H2, v_H2, ro_H2, 1.0, p_H2, u_H2, v_H2, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H2[k] = Potok_H2[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H3, 1.0, p_H3, u_H3, v_H3, ro_H3, 1.0, p_H3, u_H3, v_H3, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H3[k] = Potok_H3[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H4, 1.0, p_H4, u_H4, v_H4, ro_H4, 1.0, p_H4, u_H4, v_H4, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H4[k] = Potok_H4[k] + P[k] * S;
            }


            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u, v, ro, Q, p, u, v, P, PQ, n1, n2, dist, metod));

            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok[k] = Potok[k] + P[k] * S;
            }
            Potok[4] = Potok[4] + PQ * S;
            
        }
        else if (ii == -4)
        {
            double S = dx * 2.0;
            n1 = 0.0;
            n2 = -1.0;
            dist = dy;

            u1_polar = u;
            v1_polar = v;

            if (radius < 100)
            {
                u1_polar = u;
                v1_polar = v;
                polar_perenos(x, y, x + n1 * dx, y + n2 * dy, u1_polar, v1_polar);
            }

            if (radius > Distant)
            {
                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H1, 1.0, p_H1, u_H1, v_H1, ro_H1, 1.0, p_H1, u_H1, -v_H1, P, PQ, n1, n2, dist, metod));
                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok_H1[k] = Potok_H1[k] + P[k] * S;
                }
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H2, 1.0, p_H2, u_H2, v_H2, ro_H2, 1.0, p_H2, u_H2, -v_H2, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H2[k] = Potok_H2[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H3, 1.0, p_H3, u_H3, v_H3, ro_H3, 1.0, p_H3, u_H3, -v_H3, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H3[k] = Potok_H3[k] + P[k] * S;
            }

            tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro_H4, 1.0, p_H4, u_H4, v_H4, ro_H4, 1.0, p_H4, u_H4, -v_H4, P, PQ, n1, n2, dist, metod));
            for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
            {
                Potok_H4[k] = Potok_H4[k] + P[k] * S;
            }

            if (radius > Distant)
            {
                tmin = min(tmin, HLLC_2d_Korolkov_b_s(ro, Q, p, u1_polar, v1_polar, ro, Q, p, //
                    u1_polar, -v1_polar, P, PQ, n1, n2, dist, metod));

                for (int k = 0; k < 4; k++)  // ��������� ��� ������ � ������
                {
                    Potok[k] = Potok[k] + P[k] * S;
                }
                Potok[4] = Potok[4] + PQ * S;
            }
        }
        else
        {
            printf("Error 12438wedew4353jdyu. Ne doljni suda popadat = %d \n", ii);
        }
    }


    double U_M_H1, U_M_H2, U_M_H3, U_M_H4;
    double U_H1, U_H2, U_H3, U_H4;
    double sigma_H1, sigma_H2, sigma_H3, sigma_H4;
    double nu_H1, nu_H2, nu_H3, nu_H4;
    double q2_1, q2_2, q3;

    /// ����� ������������ �����!!!

    double ro3, p3, u3, v3, Q33;

    ro3 = ro - *T_do * (Potok[0] / Volume + ro * v / y);
    Q33 = Q - (*T_do / Volume) * Potok[4] - *T_do * Q * v / y;
    if (ro3 <= 0)
    {
        printf("Problemsssss  ro < 0! %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf\n", x, y, dx, dy, ro, p, u, v, Q);
        ro3 = 0.00001;
    }
    u3 = (ro * u - *T_do * (Potok[1] / Volume + ro * v * u / y)) / ro3;
    v3 = (ro * v - *T_do * (Potok[2] / Volume + ro * v * v / y)) / ro3;
    p3 = (((p / (ggg - 1) + ro * (u * u + v * v) * 0.5) - *T_do * (Potok[3] / Volume + //
        +v * (ggg * p / (ggg - 1) + ro * (u * u + v * v) * 0.5) / y)) - //
        0.5 * ro3 * (u3 * u3 + v3 * v3)) * (ggg - 1);
    if (p3 <= 0)
    {
        p3 = 0.000001;
    }

    Q2[index] = Q33;
    RO2[index] = ro3;
    P2[index] = p3;
    U2[index] = u3;
    V2[index] = v3;

    if (*T > tmin)
    {
        *T = tmin;
        __threadfence();
    }
    

}

