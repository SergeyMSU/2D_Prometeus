#include "hip/hip_runtime.h"
#include ""
#include "Cell.h"
#include "Header.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include "Konstruktor.h"
#include "Cuda_main.cu"
#include "Kyb.h"

#define ER_S(x) printf("Standart error in kernel.cu: kod - x\n")
#define TVD_ false
#define TVQ_ true
#define kor_Sol true

#define sss 500000000

using namespace std;

hipError_t addWithCuda(void);


int main()
{
    if (true)
    {
        // Add vectors in parallel.
        hipError_t cudaStatus = addWithCuda();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces
        // ��� ����� ������ ���� �������
        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!\n");
            return 1;
        }
    }


    return 0;
}


hipError_t addWithCuda()
{
    hipError_t cudaStatus = hipSuccess;

    Konstruktor K(30, 30, -100, 100, 100);
    K.Drobim(0, 0, 0, 50);
    K.Drobim(0, 0, 30, 80);
    K.print_konectiviti_short();
    K.print_point();
    K.print_cell();


    return cudaStatus;
}

