#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Cell.h"
#include "Header.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include "Konstruktor.h"
#include "Cuda_main.cu"
#include "Kyb.h"

#define ER_S(x) printf("Standart error in kernel.cu: kod - x\n")


using namespace std;

hipError_t addWithCuda_Gas_Din(Konstruktor& K);
__global__ void funk_time(double* T, double* T_do, double* TT, int* i);



__global__ void funk_time(double* T, double* T_do, double* TT, int* i)
{
    *T_do = *T;
    *TT = *TT + *T_do;
    *T = 10000000;
    *i = *i + 1;
    if (*i % 1000 == 0)
    {
        printf("i = %d,  TT = %lf \n", *i, *TT);
    }
    return;
}


int main()
{

    Konstruktor K(66, 56, x_min_, x_max_, y_max_);


    K.Drobim(-1500, 600, 1200);
    K.Drobim(-1000, 500, 700);
    K.Drobim(-400, 380, 350);
    K.Drobim(0, 0, 0, 300);
    K.Drobim(0, 0, 0, 160);
    K.Drobim(0, 0, 0, 80);
    K.Drobim(0, 0, 0, 40);
    K.Drobim(0, 0, 0, 20);


    K.print_cell();


    // Add vectors in parallel.

    hipError_t cudaStatus = addWithCuda_Gas_Din(K);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }




    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces
    // ��� ����� ������ ���� �������
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!\n");
        return 1;
    }
    
    return 0;
}


hipError_t addWithCuda_Gas_Din(Konstruktor& K)
{
    hipError_t cudaStatus = hipSuccess;

   

    int N = K.all_Kyb.size();          // ����� �����
    cout << "All size = " << N << endl;
    int nn = K.get_size_conektiv();    // ����� ������ (������ ������� ������)
    cout << "Connect = " << nn << endl;
    //exit(-1);


    K.initial_condition();   // ���������� ���������� ���������
    K.Download_setka("all_save_2.txt");

    int* host_sosed;
    int* dev_sosed;
    double* host_T, * host_T_do, * host_TT;
    double* dev_T, * dev_T_do, * dev_TT;
    int* host_i;
    int* dev_i;

    // ������ ������� ����������\�����

    int* dev_l, * dev_r, * Nu, * dev_N;
    double* dev_x, * dev_y;
    double* dev_Q1, * host_Q1, * dev_Q2;
    double* dev_ro1, * dev_p1, * dev_u1, * dev_v1,  * dev_ro2, * dev_p2, * dev_u2, * dev_v2;
    double* host_x, * host_y;
    double* host_ro1, * host_p1, * host_u1, * host_v1;
    int* host_l, * host_r;
    int* dev_size, * host_size;
    int met = 0;

    host_T = (double*)malloc(sizeof(double));
    host_T_do = (double*)malloc(sizeof(double));
    host_TT = (double*)malloc(sizeof(double));
    host_i = (int*)malloc(sizeof(int));
    Nu = (int*)malloc(sizeof(int));

    int NNN;
    if (N % 256 == 0)
    {
        NNN = N / 256;
    }
    else
    {
        NNN = (int)(N / 256) + 1;
    }

    host_x = new double[N];
    host_y = new double[N];
    host_ro1 = new double[N];
    host_Q1 = new double[N];
    host_size = new int[N];
    
    host_p1 = new double[N];
    host_u1 = new double[N];
    host_v1 = new double[N];
    host_l = new int[N];
    host_r = new int[N];

    *host_T = 10000000.0;
    *host_T_do = 0.00000001;
    *host_TT = 0.0;
    *host_i = 0;
    *Nu = N;

    host_sosed = new int[nn];

    // ���������� ��������
    int c = 0;
    for (auto& i : K.all_Kyb)
    {
        for (auto& j : i->sosed)
        {
            host_sosed[c] = j->number;
            c++;
        }
    }

    int ll = 0;
    for (int i = 0; i < K.all_Kyb.size(); i++)
    {
        host_size[i] = K.all_Kyb[i]->size;
        host_x[i] = K.all_Kyb[i]->x;
        host_y[i] = K.all_Kyb[i]->y;
        host_ro1[i] = K.all_Kyb[i]->ro;
        host_Q1[i] = K.all_Kyb[i]->Q;
        host_p1[i] = K.all_Kyb[i]->p;
        host_u1[i] = K.all_Kyb[i]->u;
        host_v1[i] = K.all_Kyb[i]->v;
        host_l[i] = ll;
        host_r[i] = ll + K.all_Kyb[i]->sosed.size() - 1;
        ll = ll + K.all_Kyb[i]->sosed.size();
    }

    // ��������� ������ �� �������
    if (true)
    {
        cudaStatus = hipMalloc((void**)&dev_size, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1rfef!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_x, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_y, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 2!");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&dev_ro1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Q1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Q2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }
        

        cudaStatus = hipMalloc((void**)&dev_ro2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_u1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_u2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_v1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_v2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_l, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_r, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_sosed, nn * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 2!");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&dev_T, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 3!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_T_do, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 4!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_TT, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 5!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_i, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 6!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_N, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 6!");
            goto Error;
        }
    }


    // �������� ������� � ����� �� ������
    if (true)
    {
        cudaStatus = hipMemcpy(dev_size, host_size, N * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed -vrgrvgrtg1 !");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_sosed, host_sosed, nn * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed -1 !");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_x, host_x, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_y, host_y, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_ro1, host_ro1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_Q1, host_Q1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 012!");
            goto Error;
        }
        
        cudaStatus = hipMemcpy(dev_p1, host_p1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_u1, host_u1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_v1, host_v1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_l, host_l, N * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_r, host_r, N * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_T, host_T, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 1!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_TT, host_TT, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 2!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_T_do, host_T_do, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 3!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_i, host_i, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 4!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_N, Nu, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 4!");
            goto Error;
        }
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    cout << "Start programm" << endl;
    met = 1;
    for (int i = 0; i < 100000; i = i + 2)  // ������� ����� �� ������� ������?
    {
        if (i > 550000)
        {
            met = 1;
        }
        // ��������� add() kernel �� GPU, ��������� ���������
        Cuda_main_HLLDQ << <NNN, 256 >> > (dev_N, dev_x, dev_y, dev_size,//
            dev_ro1, dev_ro2, dev_Q1, dev_Q2, dev_p1, dev_p2, dev_u1, dev_u2, dev_v1, dev_v2,//
            dev_sosed, dev_l, dev_r, dev_T, dev_T_do, i, K.DX, K.DY, met);

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 1 step\n", cudaStatus);
            goto Error;
        }

        funk_time << <1, 1 >> > (dev_T, dev_T_do, dev_TT, dev_i);
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 2 step\n", cudaStatus);
            goto Error;
        }

        Cuda_main_HLLDQ << <NNN, 256 >> > (dev_N, dev_x, dev_y, dev_size,//
            dev_ro2, dev_ro1, dev_Q2, dev_Q1, dev_p2, dev_p1, dev_u2, dev_u1, dev_v2, dev_v1,//
            dev_sosed, dev_l, dev_r, dev_T, dev_T_do, i, K.DX, K.DY, met);

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 333333\n", cudaStatus);
            goto Error;
        }

        funk_time << <1, 1 >> > (dev_T, dev_T_do, dev_TT, dev_i);
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 4444444\n", cudaStatus);
            goto Error;
        }



        if ((i % 15000000000 == 0 && i > 1))
        {
            cout << "HLL "  << endl;
            if (true)
            {
                cudaStatus = hipMemcpy(host_ro1, dev_ro1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_p1, dev_p1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_u1, dev_u1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_v1, dev_v1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_Q1, dev_Q1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
            }
            K.read_Cuda_massiv(host_ro1, host_p1, host_u1, host_v1, host_Q1);
            K.print_Tecplot();
        }
    }


    // ����������� �������� �������
    if (true)
    {
        cudaStatus = hipMemcpy(host_ro1, dev_ro1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_p1, dev_p1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_u1, dev_u1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_v1, dev_v1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_Q1, dev_Q1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
    }  

    K.read_Cuda_massiv(host_ro1, host_p1, host_u1, host_v1,  host_Q1);
    K.print_Tecplot();
    K.Save_setka("all_save_3.txt");



Error:
    hipFree(dev_sosed);
    hipFree(dev_ro1);
    hipFree(dev_ro2);
    hipFree(dev_p1);
    hipFree(dev_p2);
    hipFree(dev_u1);
    hipFree(dev_u2);
    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_T);
    hipFree(dev_i);
    hipFree(dev_T_do);
    hipFree(dev_TT);
    hipFree(dev_Q1);
    hipFree(dev_Q2);







    return cudaStatus;
}

hipError_t addWithCuda_5_komponent(Konstruktor& K)
{
    hipError_t cudaStatus = hipSuccess;



    int N = K.all_Kyb.size();          // ����� �����
    cout << "All size = " << N << endl;
    int nn = K.get_size_conektiv();    // ����� ������ (������ ������� ������)
    cout << "Connect = " << nn << endl;
    //exit(-1);


    K.initial_condition();   // ���������� ���������� ���������
    //K.Download_setka("all_save_1.txt");

    int* host_sosed;
    int* dev_sosed;
    double* host_T, * host_T_do, * host_TT;
    double* dev_T, * dev_T_do, * dev_TT;
    int* host_i;
    int* dev_i;

    // ������ ������� ����������\�����

    int* dev_l, * dev_r, * Nu, * dev_N;
    double* dev_x, * dev_y;
    double* dev_Q1, * host_Q1, * dev_Q2;
    double* dev_ro1, * dev_p1, * dev_u1, * dev_v1, * dev_ro2, * dev_p2, * dev_u2, * dev_v2;
    double* host_x, * host_y;
    double* host_ro1, * host_p1, * host_u1, * host_v1;
    double* host_ro1_H1, * host_p1_H1, * host_u1_H1, * host_v1_H1;
    double* host_ro1_H2, * host_p1_H2, * host_u1_H2, * host_v1_H2;
    double* host_ro1_H3, * host_p1_H3, * host_u1_H3, * host_v1_H3;
    double* host_ro1_H4, * host_p1_H4, * host_u1_H4, * host_v1_H4;
    double* dev_ro1_H1, * dev_p1_H1, * dev_u1_H1, * dev_v1_H1, * dev_ro2_H1, * dev_p2_H1, * dev_u2_H1, * dev_v2_H1;
    double* dev_ro1_H2, * dev_p1_H2, * dev_u1_H2, * dev_v1_H2, * dev_ro2_H2, * dev_p2_H2, * dev_u2_H2, * dev_v2_H2;
    double* dev_ro1_H3, * dev_p1_H3, * dev_u1_H3, * dev_v1_H3, * dev_ro2_H3, * dev_p2_H3, * dev_u2_H3, * dev_v2_H3;
    double* dev_ro1_H4, * dev_p1_H4, * dev_u1_H4, * dev_v1_H4, * dev_ro2_H4, * dev_p2_H4, * dev_u2_H4, * dev_v2_H4;
    int* host_l, * host_r;
    int* dev_size, * host_size;
    int met = 0;

    host_T = (double*)malloc(sizeof(double));
    host_T_do = (double*)malloc(sizeof(double));
    host_TT = (double*)malloc(sizeof(double));
    host_i = (int*)malloc(sizeof(int));
    Nu = (int*)malloc(sizeof(int));

    int NNN;
    if (N % 256 == 0)
    {
        NNN = N / 256;
    }
    else
    {
        NNN = (int)(N / 256) + 1;
    }

    host_x = new double[N];
    host_y = new double[N];
    host_Q1 = new double[N];
    host_size = new int[N];

    host_ro1 = new double[N];
    host_p1 = new double[N];
    host_u1 = new double[N];
    host_v1 = new double[N];

    host_ro1_H1 = new double[N];
    host_p1_H1 = new double[N];
    host_u1_H1 = new double[N];
    host_v1_H1 = new double[N];

    host_ro1_H2 = new double[N];
    host_p1_H2 = new double[N];
    host_u1_H2 = new double[N];
    host_v1_H2 = new double[N];

    host_ro1_H3 = new double[N];
    host_p1_H3 = new double[N];
    host_u1_H3 = new double[N];
    host_v1_H3 = new double[N];

    host_ro1_H4 = new double[N];
    host_p1_H4 = new double[N];
    host_u1_H4 = new double[N];
    host_v1_H4 = new double[N];


    host_l = new int[N];
    host_r = new int[N];

    *host_T = 10000000.0;
    *host_T_do = 0.00000001;
    *host_TT = 0.0;
    *host_i = 0;
    *Nu = N;

    host_sosed = new int[nn];

    // ���������� ��������
    int c = 0;
    for (auto& i : K.all_Kyb)
    {
        for (auto& j : i->sosed)
        {
            host_sosed[c] = j->number;
            c++;
        }
    }

    int ll = 0;
    for (int i = 0; i < K.all_Kyb.size(); i++)
    {
        host_size[i] = K.all_Kyb[i]->size;
        host_x[i] = K.all_Kyb[i]->x;
        host_y[i] = K.all_Kyb[i]->y;
        host_ro1[i] = K.all_Kyb[i]->ro;
        host_Q1[i] = K.all_Kyb[i]->Q;
        host_p1[i] = K.all_Kyb[i]->p;
        host_u1[i] = K.all_Kyb[i]->u;
        host_v1[i] = K.all_Kyb[i]->v;
        host_l[i] = ll;
        host_r[i] = ll + K.all_Kyb[i]->sosed.size() - 1;
        ll = ll + K.all_Kyb[i]->sosed.size();

        host_ro1_H1[i] = K.all_Kyb[i]->ro_H1;
        host_p1_H1[i] = K.all_Kyb[i]->p_H1;
        host_u1_H1[i] = K.all_Kyb[i]->u_H1;
        host_v1_H1[i] = K.all_Kyb[i]->v_H1;

        host_ro1_H2[i] = K.all_Kyb[i]->ro_H2;
        host_p1_H2[i] = K.all_Kyb[i]->p_H2;
        host_u1_H2[i] = K.all_Kyb[i]->u_H2;
        host_v1_H2[i] = K.all_Kyb[i]->v_H2;

        host_ro1_H3[i] = K.all_Kyb[i]->ro_H3;
        host_p1_H3[i] = K.all_Kyb[i]->p_H3;
        host_u1_H3[i] = K.all_Kyb[i]->u_H3;
        host_v1_H3[i] = K.all_Kyb[i]->v_H3;

        host_ro1_H4[i] = K.all_Kyb[i]->ro_H4;
        host_p1_H4[i] = K.all_Kyb[i]->p_H4;
        host_u1_H4[i] = K.all_Kyb[i]->u_H4;
        host_v1_H4[i] = K.all_Kyb[i]->v_H4;
    }

    // ��������� ������ �� �������
    if (true)
    {
        cudaStatus = hipMalloc((void**)&dev_size, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1rfef!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_x, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_y, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 2!");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&dev_ro1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Q1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Q2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&dev_ro2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_u1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_u2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_v1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_v2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_l, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_r, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_sosed, nn * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 2!");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&dev_T, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 3!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_T_do, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 4!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_TT, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 5!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_i, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 6!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_N, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 6!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_ro1_H1, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_ro2_H1, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_p1_H1, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_p2_H1, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_u1_H1, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_u2_H1, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_v1_H1, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_v2_H1, N * sizeof(double));

        cudaStatus = hipMalloc((void**)&dev_ro1_H2, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_ro2_H2, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_p1_H2, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_p2_H2, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_u1_H2, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_u2_H2, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_v1_H2, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_v2_H2, N * sizeof(double));

        cudaStatus = hipMalloc((void**)&dev_ro1_H3, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_ro2_H3, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_p1_H3, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_p2_H3, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_u1_H3, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_u2_H3, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_v1_H3, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_v2_H3, N * sizeof(double));

        cudaStatus = hipMalloc((void**)&dev_ro1_H4, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_ro2_H4, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_p1_H4, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_p2_H4, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_u1_H4, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_u2_H4, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_v1_H4, N * sizeof(double));
        cudaStatus = hipMalloc((void**)&dev_v2_H4, N * sizeof(double));
    }


    // �������� ������� � ����� �� ������
    if (true)
    {
        cudaStatus = hipMemcpy(dev_size, host_size, N * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed -vrgrvgrtg1 !");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_sosed, host_sosed, nn * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed -1 !");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_x, host_x, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_y, host_y, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_ro1, host_ro1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_Q1, host_Q1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 012!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_p1, host_p1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_u1, host_u1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_v1, host_v1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_l, host_l, N * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_r, host_r, N * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_T, host_T, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 1!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_TT, host_TT, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 2!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_T_do, host_T_do, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 3!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_i, host_i, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 4!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_N, Nu, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 4!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_ro1_H1, host_ro1_H1, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_p1_H1, host_p1_H1, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_u1_H1, host_u1_H1, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_v1_H1, host_v1_H1, N * sizeof(double), hipMemcpyHostToDevice);

        cudaStatus = hipMemcpy(dev_ro1_H2, host_ro1_H2, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_p1_H2, host_p1_H2, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_u1_H2, host_u1_H2, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_v1_H2, host_v1_H2, N * sizeof(double), hipMemcpyHostToDevice);

        cudaStatus = hipMemcpy(dev_ro1_H3, host_ro1_H3, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_p1_H3, host_p1_H3, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_u1_H3, host_u1_H3, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_v1_H3, host_v1_H3, N * sizeof(double), hipMemcpyHostToDevice);

        cudaStatus = hipMemcpy(dev_ro1_H4, host_ro1_H4, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_p1_H4, host_p1_H4, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_u1_H4, host_u1_H4, N * sizeof(double), hipMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_v1_H4, host_v1_H4, N * sizeof(double), hipMemcpyHostToDevice);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    cout << "Start programm" << endl;
    met = 0;
    for (int i = 0; i < 1000000; i = i + 2)  // ������� ����� �� ������� ������?
    {
        if (i > 550000)
        {
            met = 1;
        }
        // ��������� add() kernel �� GPU, ��������� ���������
        Cuda_main_HLLDQ << <NNN, 256 >> > (dev_N, dev_x, dev_y, dev_size,//
            dev_ro1, dev_ro2, dev_Q1, dev_Q2, dev_p1, dev_p2, dev_u1, dev_u2, dev_v1, dev_v2,//
            dev_sosed, dev_l, dev_r, dev_T, dev_T_do, i, K.DX, K.DY, met);

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 1 step\n", cudaStatus);
            goto Error;
        }

        funk_time << <1, 1 >> > (dev_T, dev_T_do, dev_TT, dev_i);
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 2 step\n", cudaStatus);
            goto Error;
        }

        Cuda_main_HLLDQ << <NNN, 256 >> > (dev_N, dev_x, dev_y, dev_size,//
            dev_ro2, dev_ro1, dev_Q2, dev_Q1, dev_p2, dev_p1, dev_u2, dev_u1, dev_v2, dev_v1,//
            dev_sosed, dev_l, dev_r, dev_T, dev_T_do, i, K.DX, K.DY, met);

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 333333\n", cudaStatus);
            goto Error;
        }

        funk_time << <1, 1 >> > (dev_T, dev_T_do, dev_TT, dev_i);
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 4444444\n", cudaStatus);
            goto Error;
        }



        if ((i % 15000000000 == 0 && i > 1))
        {
            cout << "HLL " << endl;
            if (true)
            {
                cudaStatus = hipMemcpy(host_ro1, dev_ro1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_p1, dev_p1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_u1, dev_u1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_v1, dev_v1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_Q1, dev_Q1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
            }
            K.read_Cuda_massiv(host_ro1, host_p1, host_u1, host_v1, host_Q1);
            K.print_Tecplot();
        }
    }


    // ����������� �������� �������
    if (true)
    {
        cudaStatus = hipMemcpy(host_ro1, dev_ro1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_p1, dev_p1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_u1, dev_u1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_v1, dev_v1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_Q1, dev_Q1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
    }

    K.read_Cuda_massiv(host_ro1, host_p1, host_u1, host_v1, host_Q1);
    K.print_Tecplot();
    K.Save_setka("all_save_1.txt");



Error:
    hipFree(dev_sosed);
    hipFree(dev_ro1);
    hipFree(dev_ro2);
    hipFree(dev_p1);
    hipFree(dev_p2);
    hipFree(dev_u1);
    hipFree(dev_u2);
    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_T);
    hipFree(dev_i);
    hipFree(dev_T_do);
    hipFree(dev_TT);
    hipFree(dev_Q1);
    hipFree(dev_Q2);







    return cudaStatus;
}

