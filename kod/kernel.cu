#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Cell.h"
#include "Header.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include "Konstruktor.h"
#include "Cuda_main.cu"
#include "Kyb.h"

#define ER_S(x) printf("Standart error in kernel.cu: kod - x\n")


using namespace std;

hipError_t addWithCuda(void);
__global__ void funk_time(double* T, double* T_do, double* TT, int* i);



__global__ void funk_time(double* T, double* T_do, double* TT, int* i)
{
    *T_do = *T;
    *TT = *TT + *T_do;
    *T = 10000000;
    *i = *i + 1;
    if (*i % 1000 == 0)
    {
        printf("i = %d,  TT = %lf \n", *i, *TT);
    }
    return;
}


int main()
{
    if (true)
    {
        // Add vectors in parallel.
        hipError_t cudaStatus = addWithCuda();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces
        // ��� ����� ������ ���� �������
        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!\n");
            return 1;
        }
    }


    return 0;
}


hipError_t addWithCuda()
{
    hipError_t cudaStatus = hipSuccess;

    Konstruktor K(30, 30, -100, 100, 100);
    K.Drobim(0, 0, 0, 50);
    K.Drobim(0, 0, 30, 80);

    int N = K.all_Kyb.size();          // ����� �����
    cout << "All size = " << N << endl;
    int nn = K.get_size_conektiv();    // ����� ������ (������ ������� ������)
    cout << "Connect = " << nn << endl;


    K.initial_condition();   // ���������� ���������� ���������


    int* host_sosed;
    int* dev_sosed;
    double* host_T, * host_T_do, * host_TT;
    double* dev_T, * dev_T_do, * dev_TT;
    int* host_i;
    int* dev_i;

    // ������ ������� ����������\�����

    int* dev_l, * dev_r, * Nu, * dev_N;
    double* dev_x, * dev_y, * dev_z;
    double* dev_dx, * dev_dy, * dev_dz;
    double* dev_Q1, * host_Q1, * dev_Q2;
    double* dev_ro1, * dev_p1, * dev_u1, * dev_v1,  * dev_ro2, * dev_p2, * dev_u2, * dev_v2;
    double* host_x, * host_y;
    double* host_ro1, * host_p1, * host_u1, * host_v1;
    int* host_l, * host_r;

    host_T = (double*)malloc(sizeof(double));
    host_T_do = (double*)malloc(sizeof(double));
    host_TT = (double*)malloc(sizeof(double));
    host_i = (int*)malloc(sizeof(int));
    Nu = (int*)malloc(sizeof(int));

    int NNN;
    if (N % 256 == 0)
    {
        NNN = N / 256;
    }
    else
    {
        NNN = (int)(N / 256) + 1;
    }

    host_x = new double[N];
    host_y = new double[N];
    host_ro1 = new double[N];
    host_Q1 = new double[N];
    
    host_p1 = new double[N];
    host_u1 = new double[N];
    host_v1 = new double[N];
    host_l = new int[N];
    host_r = new int[N];

    *host_T = 10000000.0;
    *host_T_do = 0.00000001;
    *host_TT = 0.0;
    *host_i = 0;
    *Nu = N;

    host_sosed = new int[nn];

    // ���������� ��������
    int c = 0;
    for (auto& i : K.all_Kyb)
    {
        for (auto& j : i->sosed)
        {
            host_sosed[c] = j->number;
            c++;
        }
    }

    int ll = 0;
    for (int i = 0; i < K.all_Kyb.size(); i++)
    {
        host_x[i] = K.all_Kyb[i]->x;
        host_y[i] = K.all_Kyb[i]->y;
        host_ro1[i] = K.all_Kyb[i]->ro;
        host_Q1[i] = K.all_Kyb[i]->Q;
        host_p1[i] = K.all_Kyb[i]->p;
        host_u1[i] = K.all_Kyb[i]->u;
        host_v1[i] = K.all_Kyb[i]->v;
        host_l[i] = ll;
        host_r[i] = ll + K.all_Kyb[i]->sosed.size() - 1;
        ll = ll + K.all_Kyb[i]->sosed.size();
    }

    // ��������� ������ �� �������
    if (true)
    {
        cudaStatus = hipMalloc((void**)&dev_x, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_y, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 2!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_z, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 3!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_dx, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_dy, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_dz, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_ro1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Q1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Q2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }
        

        cudaStatus = hipMalloc((void**)&dev_ro2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_u1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_u2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_v1, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_v2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_l, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_r, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 1!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_sosed, nn * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 2!");
            goto Error;
        }


        cudaStatus = hipMalloc((void**)&dev_T, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 3!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_T_do, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 4!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_TT, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 5!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_i, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 6!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_N, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed 6!");
            goto Error;
        }
    }


    // �������� ������� � ����� �� ������
    if (true)
    {
        cudaStatus = hipMemcpy(dev_sosed, host_sosed, nn * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed -1 !");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_x, host_x, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_x, host_x, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_y, host_y, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_ro1, host_ro1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_Q1, host_Q1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 012!");
            goto Error;
        }
        
        cudaStatus = hipMemcpy(dev_p1, host_p1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_u1, host_u1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_v1, host_v1, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_l, host_l, N * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_r, host_r, N * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 0!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_T, host_T, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 1!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_TT, host_TT, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 2!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_T_do, host_T_do, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 3!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_i, host_i, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 4!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_N, Nu, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed 4!");
            goto Error;
        }
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    for (int i = 0; i < 100000; i = i + 2)  // ������� ����� �� ������� ������?
    {
        // ��������� add() kernel �� GPU, ��������� ���������
        Cuda_main_HLLDQ << <NNN, 256 >> > (dev_N, dev_x, dev_y, dev_z, dev_dx, dev_dy, dev_dz,//
            dev_ro1, dev_ro2, dev_Q1, dev_Q2, dev_p1, dev_p2, dev_u1, dev_u2, dev_v1, dev_v2,//
            dev_w1, dev_w2, dev_bx1, dev_by1, dev_bz1, dev_bx2, dev_by2, dev_bz2,//
            dev_sosed, dev_l, dev_r, dev_T, dev_T_do, i, MMM, true, true, 1);

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 11111\n", cudaStatus);
            goto Error;
        }

        funk_time << <1, 1 >> > (dev_T, dev_T_do, dev_TT, dev_i);
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 222222\n", cudaStatus);
            goto Error;
        }

        Cuda_main_HLLDQ << <NNN, 256 >> > (dev_N, dev_x, dev_y, dev_z, dev_dx, dev_dy, dev_dz,//
            dev_ro2, dev_ro1, dev_Q2, dev_Q1, dev_p2, dev_p1, dev_u2, dev_u1, dev_v2, dev_v1,//
            dev_w2, dev_w1, dev_bx2, dev_by2, dev_bz2, dev_bx1, dev_by1, dev_bz1,//
            dev_sosed, dev_l, dev_r, dev_T, dev_T_do, i, MMM, true, true, 1);

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 333333\n", cudaStatus);
            goto Error;
        }

        funk_time << <1, 1 >> > (dev_T, dev_T_do, dev_TT, dev_i);
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel! 4444444\n", cudaStatus);
            goto Error;
        }



        if ((i % 15000000000 == 0 && i > 1))
        {
            cout << "HLL "  << endl;
            if (true)
            {
                cudaStatus = hipMemcpy(host_ro1, dev_ro1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_p1, dev_p1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_u1, dev_u1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_v1, dev_v1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
                cudaStatus = hipMemcpy(host_Q1, dev_Q1, N * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!  3452\n");
                    goto Error;
                }
            }
            K.read_Cuda_massiv(host_ro1, host_p1, host_u1, host_v1, host_Q1);
            K.print_Tecplot();
        }
    }


    // ����������� �������� �������
    if (true)
    {
        cudaStatus = hipMemcpy(host_ro1, dev_ro1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_p1, dev_p1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_u1, dev_u1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_v1, dev_v1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
        cudaStatus = hipMemcpy(host_Q1, dev_Q1, N * sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!  3452\n");
            goto Error;
        }
    }  

    K.read_Cuda_massiv(host_ro1, host_p1, host_u1, host_v1,  host_Q1);
    K.print_Tecplot();



Error:
    hipFree(dev_sosed);
    hipFree(dev_ro1);
    hipFree(dev_ro2);
    hipFree(dev_p1);
    hipFree(dev_p2);
    hipFree(dev_u1);
    hipFree(dev_u2);
    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_T);
    hipFree(dev_i);
    hipFree(dev_T_do);
    hipFree(dev_TT);
    hipFree(dev_Q1);
    hipFree(dev_Q2);







    return cudaStatus;
}

